#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// Copyright (c) 2017 AliceVision contributors.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

#pragma once

#include <aliceVision/depthMap/cuda/deviceCommon/device_global.cu>
#include <aliceVision/depthMap/cuda/deviceCommon/device_matrix.cu>
#include <aliceVision/depthMap/cuda/deviceCommon/device_patch_es_glob.hpp>
#include <aliceVision/depthMap/cuda/deviceCommon/device_simStat.cu>
#include <aliceVision/depthMap/cuda/planeSweeping/device_utils.cu>

#include <hip/hip_math_constants.h>

namespace aliceVision {
namespace depthMap {

__device__ void computeRotCSEpip( const CameraStructBase& rc_cam,
                                  const CameraStructBase& tc_cam,
                                  Patch& ptch )
{
    // Vector from the reference camera to the 3d point
    float3 v1 = rc_cam.C - ptch.p;
    // Vector from the target camera to the 3d point
    float3 v2 = tc_cam.C - ptch.p;
    normalize(v1);
    normalize(v2);

    // y has to be ortogonal to the epipolar plane
    // n has to be on the epipolar plane
    // x has to be on the epipolar plane

    ptch.y = cross(v1, v2);
    normalize(ptch.y);

    ptch.n = (v1 + v2) / 2.0f; // IMPORTANT !!!
    normalize(ptch.n);
    // ptch.n = sg_s_r.ZVect; //IMPORTANT !!!

    ptch.x = cross(ptch.y, ptch.n);
    normalize(ptch.x);
}

__device__ int angleBetwUnitV1andUnitV2(float3& V1, float3& V2)
{
    return (int)fabs(acos(V1.x * V2.x + V1.y * V2.y + V1.z * V2.z) / (HIP_PI_F / 180.0f));
}

/*
__device__ float getRefCamPixSize(Patch &ptch)
{
        float2 rp = project3DPoint(sg_s_r.P,ptch.p);

        float minstep=10000000.0f;
        for (int i=0;i<4;i++) {
                float2 pix = rp;
                if (i==0) {pix.x += 1.0f;};
                if (i==1) {pix.x -= 1.0f;};
                if (i==2) {pix.y += 1.0f;};
                if (i==3) {pix.y -= 1.0f;};
                float3 vect = M3x3mulV2(sg_s_r.iP,pix);
                float3 lpi = linePlaneIntersect(sg_s_r.C, vect, ptch.p, ptch.n);
                float step = dist(lpi,ptch.p);
                minstep = fminf(minstep,step);
        };

        return minstep;
}

__device__ float getTarCamPixSize(Patch &ptch)
{
        float2 tp = project3DPoint(sg_s_t.P,ptch.p);

        float minstep=10000000.0f;
        for (int i=0;i<4;i++) {
                float2 pix = tp;
                if (i==0) {pix.x += 1.0f;};
                if (i==1) {pix.x -= 1.0f;};
                if (i==2) {pix.y += 1.0f;};
                if (i==3) {pix.y -= 1.0f;};
                float3 vect = M3x3mulV2(sg_s_t.iP,pix);
                float3 lpi = linePlaneIntersect(sg_s_t.C, vect, ptch.p, ptch.n);
                float step = dist(lpi,ptch.p);
                minstep = fminf(minstep,step);
        };

        return minstep;
}

__device__ float getPatchPixSize(Patch &ptch)
{
        return fmaxf(getTarCamPixSize(ptch),getRefCamPixSize(ptch));
}
*/

__device__ void computeHomography(const CameraStructBase& rc_cam, const CameraStructBase& tc_cam, float* _H, const float3& _p, const float3& _n)
{
    // hartley zisserman second edition p.327 (13.2)
    float3 _tl = make_float3(0.0, 0.0, 0.0) - M3x3mulV3(rc_cam.R, rc_cam.C);
    float3 _tr = make_float3(0.0, 0.0, 0.0) - M3x3mulV3(tc_cam.R, tc_cam.C);

    float3 p = M3x3mulV3(rc_cam.R, (_p - rc_cam.C));
    float3 n = M3x3mulV3(rc_cam.R, _n);
    normalize(n);
    float d = -dot(n, p);

    float RrT[9];
    M3x3transpose(RrT, rc_cam.R);

    float tmpRr[9];
    M3x3mulM3x3(tmpRr, tc_cam.R, RrT);
    float3 tr = _tr - M3x3mulV3(tmpRr, _tl);

    float tmp[9];
    float tmp1[9];
    outerMultiply(tmp, tr, n / d);
    M3x3minusM3x3(tmp, tmpRr, tmp);
    M3x3mulM3x3(tmp1, tc_cam.K, tmp);
    M3x3mulM3x3(tmp, tmp1, rc_cam.iK);

    for(int i = 0; i < 9; i++)
    {
        _H[i] = tmp[i];
    }
}

/*
__device__ float compNCCbyH(const CameraStructBase& rc_cam, const CameraStructBase& tc_cam, const Patch& ptch, int wsh)
{
    float2 rpix = project3DPoint(sg_s_r.P, ptch.p);
    float2 tpix = project3DPoint(sg_s_t.P, ptch.p);

    float H[9];
    computeHomography(rc_cam, tc_cam, H, ptch.p, ptch.n);

    simStat sst = simStat();
    for(int xp = -wsh; xp <= wsh; xp++)
    {
        for(int yp = -wsh; yp <= wsh; yp++)
        {
            float2 rp;
            float2 tp;
            rp.x = rpix.x + (float)xp;
            rp.y = rpix.y + (float)yp;
            tp = V2M3x3mulV2(H, rp);

            float2 g;
            g.x = 255.0f * tex2D(rtex, rp.x + 0.5f, rp.y + 0.5f);
            g.y = 255.0f * tex2D(ttex, tp.x + 0.5f, tp.y + 0.5f);
            sst.update(g);
        }
    }
    sst.computeSim();

    return sst.sim;
}
*/

/**
 * @brief Compute Normalized Cross-Correlation
 * 
 * @param[inout] ptch
 * @param[in] wsh half-width of the similarity homography matrix (width = wsh*2+1)
 * @param[in] width image width
 * @param[in] height image height
 * @param[in] _gammaC
 * @param[in] _gammaP
 * 
 * @return similarity value
 */
__device__ float compNCCby3DptsYK( hipTextureObject_t rc_tex,
                                   hipTextureObject_t tc_tex,
                                   const CameraStructBase& rc_cam,
                                   const CameraStructBase& tc_cam,
                                   const Patch& ptch,
                                   int wsh,
                                   int rc_width, int rc_height,
                                   int tc_width, int tc_height,
                                   const float _gammaC, const float _gammaP)
{
    float3 p = ptch.p;
    float2 rp = project3DPoint(rc_cam.P, p);
    float2 tp = project3DPoint(tc_cam.P, p);

    const float dd = wsh + 2.0f; // TODO FACA
    if((rp.x < dd) || (rp.x > (float)(rc_width  - 1) - dd) ||
       (rp.y < dd) || (rp.y > (float)(rc_height - 1) - dd) ||
       (tp.x < dd) || (tp.x > (float)(tc_width  - 1) - dd) ||
       (tp.y < dd) || (tp.y > (float)(tc_height - 1) - dd))
    {
        return 1.0f;
    }

    // see CUDA_C_Programming_Guide.pdf ... E.2 pp132-133 ... adding 0.5 caises that tex2D return for point i,j exactly
    // value od I(i,j) ... it is what we want
    float4 gcr = tex2D<float4>(rc_tex, rp.x + 0.5f, rp.y + 0.5f);
    float4 gct = tex2D<float4>(tc_tex, tp.x + 0.5f, tp.y + 0.5f);

    // printf("gcr: R: %f, G: %f, B: %f, A: %f", gcr.x, gcr.y, gcr.z, gcr.w);
    // printf("gct: R: %f, G: %f, B: %f, A: %f", gct.x, gct.y, gct.z, gct.w);

    if (gcr.w == 0.0f || gct.w == 0.0f)
        return 1.0f; // if no alpha, invalid pixel from input mask

    float gammaC = _gammaC;
    // float gammaC = ((gcr.w>0)||(gct.w>0))?sigmoid(_gammaC,25.5f,20.0f,10.0f,fmaxf(gcr.w,gct.w)):_gammaC;
    // float gammaP = ((gcr.w>0)||(gct.w>0))?sigmoid(1.5,(float)(wsh+3),30.0f,20.0f,fmaxf(gcr.w,gct.w)):_gammaP;
    float gammaP = _gammaP;

    simStat sst;
    for(int yp = -wsh; yp <= wsh; yp++)
    {
        for(int xp = -wsh; xp <= wsh; xp++)
        {
            p = ptch.p + ptch.x * (float)(ptch.d * (float)xp) + ptch.y * (float)(ptch.d * (float)yp);
            float2 rp1 = project3DPoint(rc_cam.P, p);
            float2 tp1 = project3DPoint(tc_cam.P, p);

            // see CUDA_C_Programming_Guide.pdf ... E.2 pp132-133 ... adding 0.5 caises that tex2D return for point i,j
            // exactly value od I(i,j) ... it is what we want
            float4 gcr1 = tex2D<float4>(rc_tex, rp1.x + 0.5f, rp1.y + 0.5f);
            float4 gct1 = tex2D<float4>(tc_tex, tp1.x + 0.5f, tp1.y + 0.5f);

            // TODO: Does it make a difference to accurately test it for each pixel of the patch?
            // if (gcr1.w == 0.0f || gct1.w == 0.0f)
            //     continue;

            // Weighting is based on:
            //  * color difference to the center pixel of the patch:
            //    ** low value (close to 0) means that the color is different from the center pixel (ie. strongly supported surface)
            //    ** high value (close to 1) means that the color is close the center pixel (ie. uniform color)
            //  * distance in image to the center pixel of the patch:
            //    ** low value (close to 0) means that the pixel is close to the center of the patch
            //    ** high value (close to 1) means that the pixel is far from the center of the patch
            float w = CostYKfromLab(xp, yp, gcr, gcr1, gammaC, gammaP) * CostYKfromLab(xp, yp, gct, gct1, gammaC, gammaP);
            assert(w >= 0.f);
            assert(w <= 1.f);
            sst.update(gcr1.x, gct1.x, w);
        }
    }
    return sst.computeWSim();
}



/**
* @brief Compute Normalized Cross-Correlation
*
* @param[in] rc_tex
* @param[in] tc_tex3D
* @param[in] coord
* @param[in] volStepXY
* @param[in] wsh half-width of the similarity homography matrix (width = wsh*2+1)
* @param[in] gammaC
* @param[in] gammaP
*
* @return similarity value
*/
__device__ float compNCCby3DptsYK_vol(
    hipTextureObject_t rc_tex,
#ifdef PLANE_SWEEPING_PRECOMPUTED_COLORS_TEXTURE
    hipTextureObject_t tc_tex3D,
#else
    const float4* volTcamColors, const int volTcamColors_s, const int volTcamColors_p,
#endif
    const int coordX, const int coordY, const int coordZ,
    const int volDimX, const int volDimY,
    const int scale, const int volStepXY,
    const int wsh,
    const float gammaC, const float gammaP)
{
    // const dim3 coord = { coordX, coordY, coordZ };

    const float4 gcr = tex2D<float4>(rc_tex, coordX * volStepXY + 0.5f, coordY * volStepXY + 0.5f);

#ifdef PLANE_SWEEPING_PRECOMPUTED_COLORS_TEXTURE
    const float4 gct = tex3D<float4>(tc_tex3D, coordX + 0.5f, coordY + 0.5f, coordZ + 0.5f);
#else
    const float4 gct = *get3DBufferAt<float4>(volTcamColors, volTcamColors_s, volTcamColors_p, coordX, coordY, coordZ);
#endif

    // const int verbose = (int(coordX) % 200 == 0 && int(coordY) % 200 == 0 && int(coordZ) % 50 == 10);

    if( gcr.w == 0.0f || gct.w == 0.0f )
        return 1.0f; // if no alpha, invalid pixel from input mask

    simStat sst;
    for (int yp = -wsh; yp <= wsh; yp++)
    {
        float2 coord_i;
        coord_i.y = coordY + float(yp);
        if (int(coord_i.y) < 0 || int(coord_i.y) >= volDimY)
            continue;
        for (int xp = -wsh; xp <= wsh; xp++)
        {
            coord_i.x = coordX + float(xp);
            if (int(coord_i.x) < 0 || int(coord_i.x) >= volDimX)
                continue;

            const float4 gcr_i = tex2D<float4>(rc_tex, coord_i.x * volStepXY + 0.5f, coord_i.y * volStepXY + 0.5f);
            if (gcr_i.w == 0.0f)
                continue;

#ifdef PLANE_SWEEPING_PRECOMPUTED_COLORS_TEXTURE
            const float4 gct_i = tex3D<float4>(tc_tex3D, coord_i.x + 0.5f, coord_i.y + 0.5f, coordZ + 0.5f);
#else
            const float4 gct_i = *get3DBufferAt<float4>(volTcamColors, volTcamColors_s, volTcamColors_p, int(coord_i.x), int(coord_i.y), int(coordZ));
#endif
            if (gct_i.w == 0.0f)
                continue;

            // Weighting is based on:
            //  * color difference to the center pixel of the patch:
            //    ** low value (close to 0) means that the color is different from the center pixel (ie. strongly supported surface)
            //    ** high value (close to 1) means that the color is close the center pixel (ie. uniform color)
            //  * distance in image to the center pixel of the patch:
            //    ** low value (close to 0) means that the pixel is close to the center of the patch
            //    ** high value (close to 1) means that the pixel is far from the center of the patch
            const float w = CostYKfromLab(xp, yp, gcr, gcr_i, gammaC, gammaP) * CostYKfromLab(xp, yp, gct, gct_i, gammaC, gammaP);
            assert(w >= 0.f);
            assert(w <= 1.f);
            sst.update(gcr_i.x, gct_i.x, w);
        }
    }

    /*
    if (verbose)
    {
        printf("compNCCby3DptsYK_vol: coordX: %i, coordY: %i, coordZ: %i, scale: %i, volStepXY: %i\n", coordX, coordY, coordZ, scale, volStepXY);
        printf("compNCCby3DptsYK_vol: gcr: %f, %f, %f, %f\n", gcr.x, gcr.y, gcr.z, gcr.w);
        printf("compNCCby3DptsYK_vol: gct: %f, %f, %f, %f\n", gct.x, gct.y, gct.z, gct.w);
        printf("compNCCby3DptsYK_vol: sst.sim: %f\n", sst.sim);
    }*/
    return sst.computeWSim();
}

__device__ void getPixelFor3DPoint(const CameraStructBase& cam, float2& out, float3& X)
{
    float3 p = M3x4mulV3(cam.P, X);
    out = make_float2(p.x / p.z, p.y / p.z);

    if(p.z < 0.0f)
    {
        out.x = -1.0f;
        out.y = -1.0f;
    }
}

__device__ float3 get3DPointForPixelAndFrontoParellePlaneRC( const CameraStructBase& rc_cam,
                                                             const float2& pix,
                                                             float fpPlaneDepth)
{
    float3 planep = rc_cam.C + rc_cam.ZVect * fpPlaneDepth;
    float3 v = M3x3mulV2(rc_cam.iP, pix);
    normalize(v);
    return linePlaneIntersect(rc_cam.C, v, planep, rc_cam.ZVect);
}

__device__ float3 get3DPointForPixelAndFrontoParellePlaneRC( const CameraStructBase& rc_cam,
                                                             const int2& pixi,
                                                             float fpPlaneDepth)
{
    float2 pix;
    pix.x = (float)pixi.x;
    pix.y = (float)pixi.y;
    return get3DPointForPixelAndFrontoParellePlaneRC(rc_cam, pix, fpPlaneDepth);
}

__device__ float3 get3DPointForPixelAndDepthFromRC(const CameraStructBase& rc_cam, const float2& pix, float depth)
{
    float3 rpv = M3x3mulV2(rc_cam.iP, pix);
    normalize(rpv);
    return rc_cam.C + rpv * depth;
}

__device__ float3 get3DPointForPixelAndDepthFromRC(const CameraStructBase& rc_cam, const int2& pixi, float depth)
{
    float2 pix;
    pix.x = (float)pixi.x;
    pix.y = (float)pixi.y;
    return get3DPointForPixelAndDepthFromRC(rc_cam, pix, depth);
}

__device__ float3 triangulateMatchRef(const CameraStructBase& rc_cam, const CameraStructBase& tc_cam, float2& refpix, float2& tarpix)
{
    float3 refvect = M3x3mulV2(rc_cam.iP, refpix);
    normalize(refvect);
    float3 refpoint = refvect + rc_cam.C;

    float3 tarvect = M3x3mulV2(tc_cam.iP, tarpix);
    normalize(tarvect);
    float3 tarpoint = tarvect + tc_cam.C;

    float k, l;
    float3 lli1, lli2;

    lineLineIntersect(&k, &l, &lli1, &lli2, rc_cam.C, refpoint, tc_cam.C, tarpoint);

    return rc_cam.C + refvect * k;
}

__device__ float computePixSize(const CameraStructBase& cam, const float3& p)
{
    float2 rp = project3DPoint(cam.P, p);
    float2 rp1 = rp + make_float2(1.0f, 0.0f);

    float3 refvect = M3x3mulV2(cam.iP, rp1);
    normalize(refvect);
    return pointLineDistance3D(p, cam.C, refvect);
}

__device__ float refineDepthSubPixel(const float3& depths, const float3& sims)
{
    // subpixel refinement
    // subpixel refine by Stereo Matching with Color-Weighted Correlation, Hierarchical Belief Propagation, and
    // Occlusion Handling Qingxiong pami08
    // quadratic polynomial interpolation is used to approximate the cost function between three discrete depth
    // candidates: d, dA, and dB.
    // TODO: get formula back from paper as it has been lost by encoding.
    // d is the discrete depth with the minimal cost, dA ? d A 1, and dB ? d B 1. The cost function is approximated as f?x? ? ax2
    // B bx B c.
    
    float simM1 = sims.x;
    float sim = sims.y;
    float simP1 = sims.z;
    simM1 = (simM1 + 1.0f) / 2.0f;
    sim = (sim + 1.0f) / 2.0f;
    simP1 = (simP1 + 1.0f) / 2.0f;

    // sim is supposed to be the best one (so the smallest one)
    if((simM1 < sim) || (simP1 < sim))
        return depths.y; // return the input

    float dispStep = -((simP1 - simM1) / (2.0f * (simP1 + simM1 - 2.0f * sim)));

    float floatDepthM1 = depths.x;
    float floatDepthP1 = depths.z;

    //-1 : floatDepthM1
    // 0 : floatDepth
    //+1 : floatDepthP1
    // linear function fit
    // f(x)=a*x+b
    // floatDepthM1=-a+b
    // floatDepthP1= a+b
    // a = b - floatDepthM1
    // floatDepthP1=2*b-floatDepthM1
    float b = (floatDepthP1 + floatDepthM1) / 2.0f;
    float a = b - floatDepthM1;

    float interpDepth = a * dispStep + b;

    // Ensure that the interpolated value is isfinite  (i.e. neither infinite nor NaN)
    if(!isfinite(interpDepth) || interpDepth <= 0.0f)
        return depths.y; // return the input

    return interpDepth;
}

} // namespace depthMap
} // namespace aliceVision
