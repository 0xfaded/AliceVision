#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// Copyright (c) 2017 AliceVision contributors.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

namespace aliceVision {
namespace depthMap {

inline __device__ void volume_computePatch( const CameraStructBase* rc_cam_s,
                                            const CameraStructBase* tc_cam_s,
                                            patch& ptch,
                                            const float fpPlaneDepth, const int2& pix )
{
    float3 p;
    float pixSize;

    p = get3DPointForPixelAndFrontoParellePlaneRC( rc_cam_s, pix, fpPlaneDepth); // no texture use
    pixSize = computePixSize( rc_cam_s, p ); // no texture use

    ptch.p = p;
    ptch.d = pixSize;
    computeRotCSEpip( rc_cam_s, tc_cam_s, ptch, p ); // no texture use
}

__global__ void volume_init_kernel( float* volume, int volume_s, int volume_p,
                                    int volDimX, int volDimY )
{
    const int vx = blockIdx.x * blockDim.x + threadIdx.x;
    const int vy = blockIdx.y * blockDim.y + threadIdx.y;
    const int vz = blockIdx.z * blockDim.z + threadIdx.z;

    if( vx >= volDimX ) return;
    if( vy >= volDimY ) return;

    *get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz) = 9999.0f;
}

__global__ void volume_slice_kernel(
                                    hipTextureObject_t rc_tex,
                                    hipTextureObject_t tc_tex,
                                    const CameraStructBase* rc_cam_s,
                                    const CameraStructBase* tc_cam_s,
                                    const float* depths_d,
                                    const int lowestUsedDepth,
                                    const int nbDepthsToSearch,
                                    int rcWidth, int rcHeight,
                                    int tcWidth, int tcHeight,
                                    int wsh,
                                    const float gammaC, const float gammaP,
                                    float* volume_1st,
                                    int volume1st_s, int volume1st_p,
                                    float* volume_2nd,
                                    int volume2nd_s, int volume2nd_p,
                                    int volStepXY,
                                    int volDimX, int volDimY)
{
    /*
     * Note !
     * volDimX == width  / volStepXY
     * volDimY == height / volStepXY
     * width and height are needed to compute transformations,
     * volDimX and volDimY may be the number of samples, reducing memory or computation
     */

    const int vx = blockIdx.x * blockDim.x + threadIdx.x;
    const int vy = blockIdx.y * blockDim.y + threadIdx.y;
    const int vz = blockIdx.z; // * blockDim.z + threadIdx.z;

    if( vx >= volDimX || vy >= volDimY )
        return;
    if (vz >= nbDepthsToSearch)
      return;

    const int x = vx * volStepXY;
    const int y = vy * volStepXY;

    if(x >= rcWidth || y >= rcHeight)
        return;

    const int zIndex = lowestUsedDepth + vz;
    const float fpPlaneDepth = depths_d[zIndex];

    /*
    int verbose = (vx % 100 == 0 && vy % 100 == 0 && vz % 100 == 0);

    if (verbose)
    {
        printf("______________________________________\n");
        printf("volume_slice_kernel: vx: %i, vy: %i, vz: %i, x: %i, y: %i\n", vx, vy, vz, x, y);
        printf("volume_slice_kernel: volStepXY: %i, volDimX: %i, volDimY: %i\n", volStepXY, volDimX, volDimY);
        printf("volume_slice_kernel: wsh: %i\n", wsh);
        printf("volume_slice_kernel: rcWidth: %i, rcHeight: %i\n", rcWidth, rcHeight);
        printf("volume_slice_kernel: lowestUsedDepth: %i, nbDepthsToSearch: %i\n", lowestUsedDepth, nbDepthsToSearch);
        printf("volume_slice_kernel: zIndex: %i, fpPlaneDepth: %f\n", zIndex, fpPlaneDepth);
        printf("volume_slice_kernel: gammaC: %f, gammaP: %f, epipShift: %f\n", gammaC, gammaP, epipShift);
        printf("______________________________________\n");
    }
    */
    patch ptcho;
    volume_computePatch(rc_cam_s, tc_cam_s, ptcho, fpPlaneDepth, make_int2(x, y)); // no texture use

    float fsim = compNCCby3DptsYK(rc_tex, tc_tex,
                                  rc_cam_s, tc_cam_s,
                                  ptcho, wsh,
                                  rcWidth, rcHeight,
                                  tcWidth, tcHeight,
                                  gammaC, gammaP);

    const float fminVal = -1.0f;
    const float fmaxVal = 1.0f;
    fsim = (fsim - fminVal) / (fmaxVal - fminVal);
    fsim = fminf(1.0f, fmaxf(0.0f, fsim));
    fsim *= 255.0f; // Currently needed for the next step... (TODO: should be removed at some point)

    float* fsim_1st = get3DBufferAt(volume_1st, volume1st_s, volume1st_p, vx, vy, zIndex);
    float* fsim_2nd = get3DBufferAt(volume_2nd, volume2nd_s, volume2nd_p, vx, vy, zIndex);

    if (fsim < *fsim_1st)
    {
        *fsim_2nd = *fsim_1st;
        *fsim_1st = fsim;
    }
    else if (fsim < *fsim_2nd)
    {
      *fsim_2nd = fsim;
    }
}


__global__ void volume_retrieveBestZ_kernel(
  float2* bestZ, int bestZ_s,
  const float* simVolume, int simVolume_s, int simVolume_p,
  int volDimX, int volDimY, int volDimZ, int zBorder)
{
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  
  if(x >= volDimX || y >= volDimY)
    return;

  float2* outPix = get2DBufferAt(bestZ, bestZ_s, x, y);
  outPix->x = -1;
  outPix->y = 9999.0;
  for (int z = 0; z < volDimZ; ++z)
  {
    const float simAtZ = *get3DBufferAt(simVolume, simVolume_s, simVolume_p, x, y, z);
    if (simAtZ < outPix->y)
    {
      outPix->x = z;
      outPix->y = simAtZ;
    }
  }
}

/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void volume_transposeAddAvgVolume_kernel(float* volumeT, int volumeT_s, int volumeT_p,
                                                    const float* volume, int volume_s, int volume_p, int volDimX,
                                                    int volDimY, int volDimZ, int dimTrnX, int dimTrnY, int dimTrnZ,
                                                    int z, int lastN)
{
    int vx = blockIdx.x * blockDim.x + threadIdx.x;
    int vy = blockIdx.y * blockDim.y + threadIdx.y;
    int vz = z;

    if((vx >= 0) && (vx < volDimX) && (vy >= 0) && (vy < volDimY) && (vz >= 0) && (vz < volDimZ))
    {
        int v[3];
        v[0] = vx;
        v[1] = vy;
        v[2] = vz;

        int dimsTrn[3];
        dimsTrn[0] = dimTrnX;
        dimsTrn[1] = dimTrnY;
        dimsTrn[2] = dimTrnZ;

        int vTx = v[dimsTrn[0]];
        int vTy = v[dimsTrn[1]];
        int vTz = v[dimsTrn[2]];

        float* oldVal_ptr = get3DBufferAt(volumeT, volumeT_s, volumeT_p, vTx, vTy, vTz);
        float newVal = *get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz);
        float val = (*oldVal_ptr * (float)lastN + (float)newVal) / (float)(lastN + 1);

        *oldVal_ptr = val;
    }
}

template <typename T>
__global__ void volume_transposeVolume_kernel(T* volumeT, int volumeT_s, int volumeT_p, 
                                              const T* volume, int volume_s, int volume_p, 
                                              int volDimX, int volDimY, int volDimZ, 
                                              int dimTrnX, int dimTrnY, int dimTrnZ, 
                                              int z)
{
    int vx = blockIdx.x * blockDim.x + threadIdx.x;
    int vy = blockIdx.y * blockDim.y + threadIdx.y;
    int vz = z;

    if((vx >= 0) && (vx < volDimX) && (vy >= 0) && (vy < volDimY) && (vz >= 0) && (vz < volDimZ))
    {
        int v[3];
        v[0] = vx;
        v[1] = vy;
        v[2] = vz;

        int dimsTrn[3];
        dimsTrn[0] = dimTrnX;
        dimsTrn[1] = dimTrnY;
        dimsTrn[2] = dimTrnZ;

        int vTx = v[dimsTrn[0]];
        int vTy = v[dimsTrn[1]];
        int vTz = v[dimsTrn[2]];

        T* oldVal_ptr = get3DBufferAt(volumeT, volumeT_s, volumeT_p, vTx, vTy, vTz);
        T newVal = *get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz);
        *oldVal_ptr = newVal;
    }
}

template <typename T>
__global__ void volume_shiftZVolumeTempl_kernel(T* volume, int volume_s, int volume_p, int volDimX, int volDimY,
                                                int volDimZ, int vz)
{
    int vx = blockIdx.x * blockDim.x + threadIdx.x;
    int vy = blockIdx.y * blockDim.y + threadIdx.y;

    if((vx >= 0) && (vx < volDimX) && (vy >= 0) && (vy < volDimY) && (vz >= 0) && (vz < volDimZ))
    {
        T* v1_ptr = get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz);
        T* v2_ptr = get3DBufferAt(volume, volume_s, volume_p, vx, vy, volDimZ - 1 - vz);
        T v1 = *v1_ptr;
        T v2 = *v2_ptr;
        *v1_ptr = v2;
        *v2_ptr = v1;
    }
}

template <typename T>
__global__ void volume_initVolume_kernel(T* volume, int volume_s, int volume_p, int volDimX, int volDimY, int volDimZ,
                                         int vz, T cst)
{
    int vx = blockIdx.x * blockDim.x + threadIdx.x;
    int vy = blockIdx.y * blockDim.y + threadIdx.y;

    if((vx >= 0) && (vx < volDimX) && (vy >= 0) && (vy < volDimY) && (vz >= 0) && (vz < volDimZ))
    {
        T* volume_zyx = get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz);
        *volume_zyx = cst;
    }
}

__global__ void volume_updateMinXSlice_kernel(unsigned char* volume, int volume_s, int volume_p,
                                              unsigned char* xySliceBestSim, int xySliceBestSim_p,
                                              int* xySliceBestZ, int xySliceBestZ_p,
                                              int volDimX, int volDimY, int volDimZ, int vz)
{
    int vx = blockIdx.x * blockDim.x + threadIdx.x;
    int vy = blockIdx.y * blockDim.y + threadIdx.y;

    if( ( vx >= volDimX ) || ( vy >= volDimY ) || ( vz >= volDimZ ) || ( vz < 0 ) ) return;

    unsigned char sim = *get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz);
    BufPtr<unsigned char> xySliceBest( xySliceBestSim, xySliceBestSim_p );
    unsigned char actSim_ptr = xySliceBest.at(vx, vy);
    if((sim < actSim_ptr) || (vz == 0))
    {
        xySliceBest                              .at(vx,vy) = sim;
        BufPtr<int>(xySliceBestZ, xySliceBestZ_p).at(vx,vy) = vz;
    }
}

template <typename T1, typename T2>
__global__ void volume_getVolumeXYSliceAtZ_kernel(T1* xySlice, int xySlice_p, T2* volume, int volume_s, int volume_p,
                                                  int volDimX, int volDimY, int volDimZ, int vz)
{
    int vx = blockIdx.x * blockDim.x + threadIdx.x;
    int vy = blockIdx.y * blockDim.y + threadIdx.y;

    if((vx >= 0) && (vx < volDimX) && (vy >= 0) && (vy < volDimY) && (vz >= 0) && (vz < volDimZ))
    {
        T2* volume_zyx = get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz);
        T1* xySlice_yx = get2DBufferAt(xySlice, xySlice_p, vx, vy);
        *xySlice_yx = (T1)(*volume_zyx);
    }
}

__global__ void volume_agregateCostVolumeAtZ_kernel(float* volume, int volume_s, int volume_p,
                                                    float* xsliceBestInColCst, int volDimX, int volDimY,
                                                    int volDimZ, int vz, float P1, float P2,
                                                    bool transfer)
{
    int vx = blockIdx.x * blockDim.x + threadIdx.x;
    int vy = blockIdx.y * blockDim.y + threadIdx.y;

    if((vx >= 0) && (vx < volDimX) && (vy >= 0) && (vy < volDimY) && (vz >= 0) && (vz < volDimZ))
    {
        float* sim_ptr = get3DBufferAt(volume, volume_s, volume_p, vx, vy, vz);
        float sim = *sim_ptr;
        float pathCost = (transfer == true) ? sim : 255.0f;

        if((vz >= 1) && (vy >= 1) && (vy < volDimY - 1))
        {
            float bestCostM = xsliceBestInColCst[vx];
            float pathCostMDM1 = volume[(vz - 1) * volume_s + (vy - 1) * volume_p + vx];
            float pathCostMD = volume[(vz - 1) * volume_s + (vy + 0) * volume_p + vx];
            float pathCostMDP1 = volume[(vz - 1) * volume_s + (vy + 1) * volume_p + vx];
            pathCost = sim + multi_fminf(pathCostMD, pathCostMDM1 + P1, pathCostMDP1 + P1, bestCostM + P2) - bestCostM;
            pathCost = pathCost;
        }

        *sim_ptr = pathCost;
    }
}

__global__ void volume_computeBestXSlice_kernel(float* xySlice, int xySlice_p, float* xsliceBestInColCst, int volDimX, int volDimY)
{
    int vx = blockIdx.x * blockDim.x + threadIdx.x;

    if((vx >= 0) && (vx < volDimX))
    {
        float bestCst = *get2DBufferAt(xySlice, xySlice_p, vx, 0);

        for(int vy = 0; vy < volDimY; vy++)
        {
            float cst = *get2DBufferAt(xySlice, xySlice_p, vx, vy);
            bestCst = cst < bestCst ? cst : bestCst;
        }
        xsliceBestInColCst[vx] = bestCst;
    }
}

/**
 * @param[inout] xySliceForZ input similarity plane
 * @param[in] xySliceForZM1
 * @param[in] xSliceBestInColCst
 * @param[out] volSimT output similarity volume
 */
__global__ void volume_agregateCostVolumeAtZinSlices_kernel(hipTextureObject_t rc_tex,
                                                            float* xySliceForZ, int xySliceForZ_p,
                                                            const float* xySliceForZM1, int xySliceForZM1_p,
                                                            const float* xSliceBestInColSimForZM1,
                                                            float* volSimT, int volSimT_s, int volSimT_p,
                                                            int volDimX, int volDimY, int volDimZ, 
                                                            int vz, unsigned int _P1, unsigned int _P2,
                                                            int dimTrnX, bool doInvZ)
{
    int vx = blockIdx.x * blockDim.x + threadIdx.x;
    int vy = blockIdx.y * blockDim.y + threadIdx.y;

    if((vx >= 0) && (vx < volDimX) && (vy >= 0) && (vy < volDimY) && (vz >= 0) && (vz < volDimZ))
    {
        float* sim_yx = get2DBufferAt(xySliceForZ, xySliceForZ_p, vx, vy);
        float sim = *sim_yx;
        float pathCost = 255.0f;

        if((vz >= 1) && (vy >= 1) && (vy < volDimY - 1))
        {
            int z = doInvZ ? volDimZ - vz : vz;
            int z1 = doInvZ ? z + 1 : z - 1; // M1
            int imX0 = (dimTrnX == 0) ? vx : z; // current
            int imY0 = (dimTrnX == 0) ?  z : vx;
            int imX1 = (dimTrnX == 0) ? vx : z1; // M1
            int imY1 = (dimTrnX == 0) ? z1 : vx;
            float4 gcr0 = 255.0f * tex2D<float4>(rc_tex, (float)imX0 + 0.5f, (float)imY0 + 0.5f);
            float4 gcr1 = 255.0f * tex2D<float4>(rc_tex, (float)imX1 + 0.5f, (float)imY1 + 0.5f);
            float deltaC = Euclidean3(gcr0, gcr1);
            // unsigned int P1 = (unsigned int)sigmoid(5.0f,20.0f,60.0f,10.0f,deltaC);
            float P1 = _P1;
            // 15.0 + (255.0 - 15.0) * (1.0 / (1.0 + exp(10.0 * ((x - 20.) / 80.))))
            float P2 = sigmoid(15.0f, 255.0f, 80.0f, 20.0f, deltaC);
            // float P2 = _P2;

            float bestCostInColM1 = xSliceBestInColSimForZM1[vx];
            float pathCostMDM1 = *get2DBufferAt(xySliceForZM1, xySliceForZM1_p, vx, vy - 1); // M1: minus 1 over depths
            float pathCostMD   = *get2DBufferAt(xySliceForZM1, xySliceForZM1_p, vx, vy);
            float pathCostMDP1 = *get2DBufferAt(xySliceForZM1, xySliceForZM1_p, vx, vy + 1); // P1: plus 1 over depths
            float minCost = multi_fminf(pathCostMD, pathCostMDM1 + P1, pathCostMDP1 + P1, bestCostInColM1 + P2);

            // if 'pathCostMD' is the minimal value of the depth
            pathCost = sim + minCost - bestCostInColM1;
        }
        float* volume_zyx = get3DBufferAt(volSimT, volSimT_s, volSimT_p, vx, vy, vz);
        *volume_zyx = pathCost;
        *sim_yx = pathCost;
    }
}

} // namespace depthMap
} // namespace aliceVision
