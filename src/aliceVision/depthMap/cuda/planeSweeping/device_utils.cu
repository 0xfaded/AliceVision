#include "hip/hip_runtime.h"
// This file is part of the AliceVision project.
// Copyright (c) 2017 AliceVision contributors.
// This Source Code Form is subject to the terms of the Mozilla Public License,
// v. 2.0. If a copy of the MPL was not distributed with this file,
// You can obtain one at https://mozilla.org/MPL/2.0/.

#pragma once

#include <aliceVision/depthMap/cuda/planeSweeping/device_utils.h>


namespace aliceVision {
namespace depthMap {

/**
* @brief
* @param[int] ptr
* @param[int] pitch raw length of a line in bytes
* @param[int] x
* @param[int] y
* @return
*/
template <typename T>
__device__ T* get2DBufferAt(T* ptr, int pitch, int x, int y)
{
    return &(BufPtr<T>(ptr,pitch).at(x,y));
}

/**
* @brief
* @param[int] ptr
* @param[int] spitch raw length of a 2D array in bytes
* @param[int] pitch raw length of a line in bytes
* @param[int] x
* @param[int] y
* @return
*/
template <typename T>
__device__ T* get3DBufferAt(T* ptr, int spitch, int pitch, int x, int y, int z)
{
    return ((T*)(((char*)ptr) + z * spitch + y * pitch)) + x;
}

__device__ float multi_fminf(float a, float b, float c)
{
  return fminf(fminf(a, b), c);
}

__device__ float multi_fminf(float a, float b, float c, float d)
{
  return fminf(fminf(fminf(a, b), c), d);
}


} // namespace depthMap
} // namespace aliceVision
