#include "hip/hip_runtime.h"
#ifndef HOST_CODE_CU
#define HOST_CODE_CU

#include "cudatemplates/copy.hpp"
#include "cudatemplates/devicememorypitched.hpp"
#include "cudatemplates/hostmemoryheap.hpp"

#include "../common_gpu_cpu_structures.h"

#include "device_code.cu"
#include "device_global.cu"
#include "device_matrix.cu"
#include "device_simStat.cu"

//-----------------------------------------------------------------------------
// Macro for checking cuda errors
#define CHECK_CUDA_ERROR()                                                                                             \
    if(hipError_t err = hipGetLastError())                                                                           \
    {                                                                                                                  \
        fprintf(stderr, "\n\nCUDAError: %s\n", hipGetErrorString(err));                                               \
        fprintf(stderr, "  file:       %s\n", __FILE__);                                                               \
        fprintf(stderr, "  function:   %s\n", __FUNCTION__);                                                           \
        fprintf(stderr, "  line:       %d\n\n", __LINE__);                                                             \
        throw ::Cuda::Error(__FILE__, __LINE__, __PRETTY_FUNCTION__, (int)err, 0);                                     \
    }

//-----------------------------------------------------------------------------
// Round a / b to nearest higher integer value.
inline unsigned int divUp(unsigned int a, unsigned int b)
{
    return (a % b != 0) ? (a / b + 1) : (a / b);
}

void refineConfDeviceAllocate(int ncams, Cuda::HostMemoryHeap<unsigned char, 2>** imgs_hmh, int width, int height)
{
    ///////////////////////////////////////////////////////////////////////////////
    // setup textures parameters
    ref_rtex.filterMode = hipFilterModeLinear;
    ref_rtex.normalized = false;
    ref_ttex.filterMode = hipFilterModeLinear;
    ref_ttex.normalized = false;

    ///////////////////////////////////////////////////////////////////////////////
    // copy textures to the device
    ref_triangles_texs_arr = new Cuda::Array<unsigned char, 2>*[ncams];
    for(int c = 0; c < ncams; c++)
    {
        ref_triangles_texs_arr[c] = new Cuda::Array<unsigned char, 2>(*imgs_hmh[c]);
    };

    hipDeviceSynchronize();
}

void refineConfDeviceDeallocate(int ncams)
{
    for(int c = 0; c < ncams; c++)
    {
        delete ref_triangles_texs_arr[c];
    };
    delete[] ref_triangles_texs_arr;
}

__host__ void refine_init_reference_camera_matrices(float* _P, float* _iP, float* _R, float* _iR, float* _K, float* _iK,
                                                    float* _C)
{
    hipMemcpyToSymbol(HIP_SYMBOL(rP), _P, sizeof(float) * 3 * 4);
    hipMemcpyToSymbol(HIP_SYMBOL(riP), _iP, sizeof(float) * 3 * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(rR), _R, sizeof(float) * 3 * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(riR), _iR, sizeof(float) * 3 * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(rK), _K, sizeof(float) * 3 * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(riK), _iK, sizeof(float) * 3 * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(rC), _C, sizeof(float) * 3);
}

__host__ void refine_init_target_camera_matrices(float* _P, float* _iP, float* _R, float* _iR, float* _K, float* _iK,
                                                 float* _C)
{
    hipMemcpyToSymbol(HIP_SYMBOL(tP), _P, sizeof(float) * 3 * 4);
    hipMemcpyToSymbol(HIP_SYMBOL(tiP), _iP, sizeof(float) * 3 * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(tR), _R, sizeof(float) * 3 * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(tiR), _iR, sizeof(float) * 3 * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(tK), _K, sizeof(float) * 3 * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(tiK), _iK, sizeof(float) * 3 * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(tC), _C, sizeof(float) * 3);
}

void refineConf(Cuda::HostMemoryHeap<float3, 2>& t_hmh, // 3dtriangles
                Cuda::HostMemoryHeap<float, 2>* s_hmh,  // similarity or confidence
                int rc, int tc, int ntris, cameraStruct& rccam, cameraStruct& tccam)
{
    assert(rc > -1);
    assert(tc > -1);

    // printf("%i\n",ntris);

    ///////////////////////////////////////////////////////////////////////////////
    // setup block and grid
    int block_size = 64;
    dim3 block(block_size, 1, 1);
    dim3 grid(divUp(ntris, block_size), 1, 1);

    ///////////////////////////////////////////////////////////////////////////////
    // copy data to the device memory
    // Cuda::DeviceMemoryPitched<float2,2> t_dmp(t_hmh);
    // Cuda::DeviceMemoryPitched<float,2>  s_dmp(Cuda::Size<2>(ntris, 1));

    float3* ptrA;
    CUDA_CHECK(hipMalloc((void**)&ptrA, ntris * sizeof(float3)));
    CUDA_CHECK(hipMemcpy(ptrA, &t_hmh.getBuffer()[0 * ntris], ntris * sizeof(float3), hipMemcpyHostToDevice));

    float3* ptrB;
    CUDA_CHECK(hipMalloc((void**)&ptrB, ntris * sizeof(float3)));
    CUDA_CHECK(hipMemcpy(ptrB, &t_hmh.getBuffer()[1 * ntris], ntris * sizeof(float3), hipMemcpyHostToDevice));

    float3* ptrC;
    CUDA_CHECK(hipMalloc((void**)&ptrC, ntris * sizeof(float3)));
    CUDA_CHECK(hipMemcpy(ptrC, &t_hmh.getBuffer()[2 * ntris], ntris * sizeof(float3), hipMemcpyHostToDevice));

    float* ptsim;
    CUDA_CHECK(hipMalloc((void**)&ptsim, ntris * sizeof(float)));

    ///////////////////////////////////////////////////////////////////////////////
    // bind textures
    hipBindTextureToArray(ref_rtex, ref_triangles_texs_arr[rc]->getArray(), hipCreateChannelDesc<unsigned char>());
    hipBindTextureToArray(ref_ttex, ref_triangles_texs_arr[tc]->getArray(), hipCreateChannelDesc<unsigned char>());

    ///////////////////////////////////////////////////////////////////////////////
    // setup reference and target matrices to the constant memory
    refine_init_reference_camera_matrices(rccam.P, rccam.iP, rccam.R, rccam.iR, rccam.K, rccam.iK, rccam.C);
    refine_init_target_camera_matrices(tccam.P, tccam.iP, tccam.R, tccam.iR, tccam.K, tccam.iK, tccam.C);

    ///////////////////////////////////////////////////////////////////////////////
    // compute similarity rotation map
    triangles_kernel<<<grid, block>>>(ptrA, ptrB, ptrC, ptsim, ntris);
    CHECK_CUDA_ERROR();

    ///////////////////////////////////////////////////////////////////////////////
    // copy(*s_hmh,s_dmp);
    CUDA_CHECK(hipMemcpy(s_hmh->getBuffer(), ptsim, ntris * sizeof(float), hipMemcpyDeviceToHost));

    ///////////////////////////////////////////////////////////////////////////////
    // unbind textures
    hipUnbindTexture(ref_rtex);
    hipUnbindTexture(ref_ttex);

    hipFree(ptrA);
    hipFree(ptrB);
    hipFree(ptrC);
    hipFree(ptsim);

    ///////////////////////////////////////////////////////////////////////////////
    hipDeviceSynchronize();
}

void refineConfPatches(Cuda::HostMemoryHeap<float3, 2>& p_hmh, // 3dpoints
                       Cuda::HostMemoryHeap<float, 2>* s_hmh,  // similarity or confidence
                       int rc, int tc, int npts, cameraStruct& rccam, cameraStruct& tccam)
{
    assert(rc > -1);
    assert(tc > -1);

    // printf("%i\n",ntris);

    ///////////////////////////////////////////////////////////////////////////////
    // setup block and grid
    int block_size = 64;
    dim3 block(block_size, 1, 1);
    dim3 grid(divUp(npts, block_size), 1, 1);

    ///////////////////////////////////////////////////////////////////////////////
    // copy data to the device memory
    // Cuda::DeviceMemoryPitched<float2,2> t_dmp(t_hmh);
    // Cuda::DeviceMemoryPitched<float,2>  s_dmp(Cuda::Size<2>(ntris, 1));

    float3* ptr;
    CUDA_CHECK(hipMalloc((void**)&ptr, npts * sizeof(float3)));
    CUDA_CHECK(hipMemcpy(ptr, &p_hmh.getBuffer()[0 * npts], npts * sizeof(float3), hipMemcpyHostToDevice));

    float* ptsim;
    CUDA_CHECK(hipMalloc((void**)&ptsim, npts * sizeof(float)));

    ///////////////////////////////////////////////////////////////////////////////
    // bind textures
    hipBindTextureToArray(ref_rtex, ref_triangles_texs_arr[rc]->getArray(), hipCreateChannelDesc<unsigned char>());
    hipBindTextureToArray(ref_ttex, ref_triangles_texs_arr[tc]->getArray(), hipCreateChannelDesc<unsigned char>());

    ///////////////////////////////////////////////////////////////////////////////
    // setup reference and target matrices to the constant memory
    refine_init_reference_camera_matrices(rccam.P, rccam.iP, rccam.R, rccam.iR, rccam.K, rccam.iK, rccam.C);
    refine_init_target_camera_matrices(tccam.P, tccam.iP, tccam.R, tccam.iR, tccam.K, tccam.iK, tccam.C);

    ///////////////////////////////////////////////////////////////////////////////
    // compute similarity rotation map
    patches_kernel<<<grid, block>>>(ptr, ptsim, npts);
    CHECK_CUDA_ERROR();

    ///////////////////////////////////////////////////////////////////////////////
    // copy(*s_hmh,s_dmp);
    CUDA_CHECK(hipMemcpy(s_hmh->getBuffer(), ptsim, npts * sizeof(float), hipMemcpyDeviceToHost));

    ///////////////////////////////////////////////////////////////////////////////
    // unbind textures
    hipUnbindTexture(ref_rtex);
    hipUnbindTexture(ref_ttex);

    hipFree(ptr);
    hipFree(ptsim);

    ///////////////////////////////////////////////////////////////////////////////
    hipDeviceSynchronize();
}

#endif // HOST_CODE_CU
